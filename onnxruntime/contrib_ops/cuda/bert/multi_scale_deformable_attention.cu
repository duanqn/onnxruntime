#include <hip/hip_runtime.h>

#include <stdexcept>

#define CUDA_KERNEL_LOOP(i, n)                          \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;   \
      i < (n);                                          \
      i += blockDim.x * gridDim.x)

__device__ float ms_deform_attn_im2col_bilinear(const float* &bottom_data,
                                                   const int &height, const int &width, const int &nheads, const int &channels,
                                                   const float &h, const float &w, const int &m, const int &c)
{
  const int h_low = floor(h);
  const int w_low = floor(w);
  const int h_high = h_low + 1;
  const int w_high = w_low + 1;

  const float lh = h - h_low;
  const float lw = w - w_low;
  const float hh = 1 - lh, hw = 1 - lw;

  const int w_stride = nheads * channels;
  const int h_stride = width * w_stride;
  const int h_low_ptr_offset = h_low * h_stride;
  const int h_high_ptr_offset = h_low_ptr_offset + h_stride;
  const int w_low_ptr_offset = w_low * w_stride;
  const int w_high_ptr_offset = w_low_ptr_offset + w_stride;
  const int base_ptr = m * channels + c;

  float v1 = 0;
  if (h_low >= 0 && w_low >= 0)
  {
    const int ptr1 = h_low_ptr_offset + w_low_ptr_offset + base_ptr;
    v1 = bottom_data[ptr1];
  }
  float v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
  {
    const int ptr2 = h_low_ptr_offset + w_high_ptr_offset + base_ptr;
    v2 = bottom_data[ptr2];
  }
  float v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
  {
    const int ptr3 = h_high_ptr_offset + w_low_ptr_offset + base_ptr;
    v3 = bottom_data[ptr3];
  }
  float v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
  {
    const int ptr4 = h_high_ptr_offset + w_high_ptr_offset + base_ptr;
    v4 = bottom_data[ptr4];
  }

  const float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  const float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

__global__ void ms_deformable_im2col_gpu_kernel(const int n,
                                                const float *data_value,
                                                const int64_t *data_spatial_shapes,
                                                const float *data_sampling_loc,
                                                const float *data_attn_weight,
                                                const int batch_size,
                                                const int spatial_size,
                                                const int num_heads,
                                                const int channels,
                                                const int num_levels,
                                                const int num_query,
                                                const int num_point,
                                                float *data_col)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    int _temp = index;
    const int c_col = _temp % channels;
    _temp /= channels;
    const int sampling_index = _temp;
    const int m_col = _temp % num_heads;
    _temp /= num_heads;
    const int q_col = _temp % num_query;
    _temp /= num_query;
    const int b_col = _temp;

    float *data_col_ptr = data_col + index;
    int data_weight_ptr = sampling_index * num_levels * num_point;
    int data_loc_w_ptr = data_weight_ptr << 1;
    const int qid_stride = num_heads * channels;
    const int data_value_ptr_init_offset = b_col * spatial_size * qid_stride;
    float col = 0;

    int level_start_id = 0;
    for (int l_col=0; l_col < num_levels; ++l_col)
    {
      const int spatial_h_ptr = l_col << 1;
      const int spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const float *data_value_ptr = data_value + (data_value_ptr_init_offset + level_start_id * qid_stride);
      for (int p_col=0; p_col < num_point; ++p_col)
      {
        const float loc_w = data_sampling_loc[data_loc_w_ptr];
        const float loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const float weight = data_attn_weight[data_weight_ptr];

        // Note(qiduan): scaling is removed
        const float h_im = loc_h - 0.5;
        const float w_im = loc_w - 0.5;

        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w)
        {
          col += ms_deform_attn_im2col_bilinear(data_value_ptr, spatial_h, spatial_w, num_heads, channels, h_im, w_im, m_col, c_col) * weight;
        }

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
      }
      level_start_id += spatial_h * spatial_w;
    }
    *data_col_ptr = col;
  }
}

void ms_deformable_im2col_gpu_kernel_wrapper(
  const float *data_value,
  const int64_t *data_spatial_shapes,
  const float *data_sampling_loc,
  const float *data_attn_weight,
  const int batch_size,
  const int spatial_size,
  const int num_heads,
  const int channels,
  const int num_levels,
  const int num_query,
  const int num_point,
  float *output)
{
  hipError_t err;
  int gpuCount = 0;
  if((err = hipGetDeviceCount(&gpuCount)) != hipSuccess){
    throw std::runtime_error(hipGetErrorString(err));
  }

  if(gpuCount <= 0){
    throw std::runtime_error("No CUDA compatible GPU available");
  }

  const int deviceID = 0;
  // Run on device 0
  if((err = hipSetDevice(deviceID)) != hipSuccess){
    throw std::runtime_error(hipGetErrorString(err));
  }

  hipDeviceProp_t property;
  if((err = hipGetDeviceProperties(&property, deviceID)) != hipSuccess){
    throw std::runtime_error(hipGetErrorString(err));
  }

  int num_output = batch_size * num_query * num_heads * channels;
  int num_threads = std::min(1024, property.maxThreadsPerBlock);
  int blocks = (num_output + num_threads - 1) / num_threads;

  hipStream_t stream;
  if((err = hipStreamCreate(&stream)) != hipSuccess){
    throw std::runtime_error(hipGetErrorString(err));
  }

  ms_deformable_im2col_gpu_kernel<<<blocks, num_threads, num_threads*3*sizeof(float), stream>>>(
    num_output,
    data_value,
    data_spatial_shapes,
    data_sampling_loc,
    data_attn_weight,
    batch_size,
    spatial_size,
    num_heads,
    channels,
    num_levels,
    num_query,
    num_point,
    output
  );

  if((err = hipStreamDestroy(stream)) != hipSuccess){
    throw std::runtime_error(hipGetErrorString(err));
  }
}
