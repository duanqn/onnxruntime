#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void ms_deformable_im2col_gpu_kernel(const int n,
                                                const float *data_value,
                                                const int64_t *data_spatial_shapes,
                                                const int64_t *data_level_start_index,
                                                const float *data_sampling_loc,
                                                const float *data_attn_weight,
                                                const int batch_size,
                                                const int spatial_size,
                                                const int num_heads,
                                                const int channels,
                                                const int num_levels,
                                                const int num_query,
                                                const int num_point,
                                                float *data_col)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    int _temp = index;
    const int c_col = _temp % channels;
    _temp /= channels;
    const int sampling_index = _temp;
    const int m_col = _temp % num_heads;
    _temp /= num_heads;
    const int q_col = _temp % num_query;
    _temp /= num_query;
    const int b_col = _temp;

    float *data_col_ptr = data_col + index;
    int data_weight_ptr = sampling_index * num_levels * num_point;
    int data_loc_w_ptr = data_weight_ptr << 1;
    const int qid_stride = num_heads * channels;
    const int data_value_ptr_init_offset = b_col * spatial_size * qid_stride;
    float col = 0;

    for (int l_col=0; l_col < num_levels; ++l_col)
    {
      const int level_start_id = data_level_start_index[l_col];
      const int spatial_h_ptr = l_col << 1;
      const int spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const float *data_value_ptr = data_value + (data_value_ptr_init_offset + level_start_id * qid_stride);
      for (int p_col=0; p_col < num_point; ++p_col)
      {
        const float loc_w = data_sampling_loc[data_loc_w_ptr];
        const float loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const float weight = data_attn_weight[data_weight_ptr];

        const float h_im = loc_h * spatial_h - 0.5;
        const float w_im = loc_w * spatial_w - 0.5;

        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w)
        {
          col += ms_deform_attn_im2col_bilinear(data_value_ptr, spatial_h, spatial_w, num_heads, channels, h_im, w_im, m_col, c_col) * weight;
        }

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
      }
    }
    *data_col_ptr = col;
  }
}

__device__ float ms_deform_attn_im2col_bilinear(const float* &bottom_data,
                                                   const int &height, const int &width, const int &nheads, const int &channels,
                                                   const float &h, const float &w, const int &m, const int &c)
{
  const int h_low = floor(h);
  const int w_low = floor(w);
  const int h_high = h_low + 1;
  const int w_high = w_low + 1;

  const float lh = h - h_low;
  const float lw = w - w_low;
  const float hh = 1 - lh, hw = 1 - lw;

  const int w_stride = nheads * channels;
  const int h_stride = width * w_stride;
  const int h_low_ptr_offset = h_low * h_stride;
  const int h_high_ptr_offset = h_low_ptr_offset + h_stride;
  const int w_low_ptr_offset = w_low * w_stride;
  const int w_high_ptr_offset = w_low_ptr_offset + w_stride;
  const int base_ptr = m * channels + c;

  float v1 = 0;
  if (h_low >= 0 && w_low >= 0)
  {
    const int ptr1 = h_low_ptr_offset + w_low_ptr_offset + base_ptr;
    v1 = bottom_data[ptr1];
  }
  float v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
  {
    const int ptr2 = h_low_ptr_offset + w_high_ptr_offset + base_ptr;
    v2 = bottom_data[ptr2];
  }
  float v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
  {
    const int ptr3 = h_high_ptr_offset + w_low_ptr_offset + base_ptr;
    v3 = bottom_data[ptr3];
  }
  float v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
  {
    const int ptr4 = h_high_ptr_offset + w_high_ptr_offset + base_ptr;
    v4 = bottom_data[ptr4];
  }

  const float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  const float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}
