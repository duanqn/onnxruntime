#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>

#include <stdexcept>

#include "core/framework/stream_handles.h"

#define CUDA_KERNEL_LOOP(i, n)                          \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;   \
      i < (n);                                          \
      i += blockDim.x * gridDim.x)

__device__ float ms_deform_attn_im2col_bilinear(const float* &bottom_data,
                                                   const int &height, const int &width, const int &nheads, const int &channels,
                                                   const float &h, const float &w, const int &m, const int &c)
{
  const int h_low = floor(h);
  const int w_low = floor(w);
  const int h_high = h_low + 1;
  const int w_high = w_low + 1;

  const float lh = h - h_low;
  const float lw = w - w_low;
  const float hh = 1 - lh, hw = 1 - lw;

  const int w_stride = nheads * channels;
  const int h_stride = width * w_stride;
  const int h_low_ptr_offset = h_low * h_stride;
  const int h_high_ptr_offset = h_low_ptr_offset + h_stride;
  const int w_low_ptr_offset = w_low * w_stride;
  const int w_high_ptr_offset = w_low_ptr_offset + w_stride;
  const int base_ptr = m * channels + c;

  float v1 = 0;
  if (h_low >= 0 && w_low >= 0)
  {
    const int ptr1 = h_low_ptr_offset + w_low_ptr_offset + base_ptr;
    v1 = bottom_data[ptr1];
  }
  float v2 = 0;
  if (h_low >= 0 && w_high <= width - 1)
  {
    const int ptr2 = h_low_ptr_offset + w_high_ptr_offset + base_ptr;
    v2 = bottom_data[ptr2];
  }
  float v3 = 0;
  if (h_high <= height - 1 && w_low >= 0)
  {
    const int ptr3 = h_high_ptr_offset + w_low_ptr_offset + base_ptr;
    v3 = bottom_data[ptr3];
  }
  float v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1)
  {
    const int ptr4 = h_high_ptr_offset + w_high_ptr_offset + base_ptr;
    v4 = bottom_data[ptr4];
  }

  const float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  const float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

__global__ void ms_deformable_im2col_gpu_kernel(const int n,
                                                const float *data_value,
                                                const int64_t *data_spatial_shapes,
                                                const float *data_sampling_loc,
                                                const float *data_reference_points,
                                                const float *data_attn_weight,
                                                const int B,
                                                const int S,
                                                const int M,
                                                const int C,
                                                const int L,
                                                const int Q,
                                                const int P,
                                                float *data_col)
{
  assert(n == B * Q * M * C);
  CUDA_KERNEL_LOOP(index, n)
  {
    int _temp = index;
    const int c = _temp % C;
    _temp /= C;
    const int m = _temp % M;
    _temp /= M;
    const int q = _temp % Q;
    _temp /= Q;
    const int b = _temp;
    assert(b == 0);
    assert(B == 1);
    assert(L == 4);

    float *data_col_ptr = data_col + index;
    // data_attn_weight shape: [B, L, Q, M, P]
    int data_weight_ptr = 0;
    // data_sampling_loc shape: [B, L, Q, M, P, 2]
    int data_loc_w_ptr = 0;
    int data_reference_ptr = (b * L * Q + 0 * Q + q) << 1;
    float res = 0;

    int level_start_id = 0;
    for (int l = 0; l < L; ++l)
    {
      data_weight_ptr = ((b * L + l) * Q + q) * M + m * P;  // at p=0
      data_loc_w_ptr = data_weight_ptr << 1;
      const int spatial_h = data_spatial_shapes[l * 2];
      const int spatial_w = data_spatial_shapes[l * 2 + 1];
      const float *data_value_ptr = data_value + (b * S + level_start_id) * M * C;
      for (int p = 0; p < P; ++p)
      {
        const float loc_w = data_sampling_loc[data_loc_w_ptr] + data_reference_points[data_reference_ptr];
        const float loc_h = data_sampling_loc[data_loc_w_ptr + 1] + data_reference_points[data_reference_ptr + 1];
        const float weight = data_attn_weight[data_weight_ptr];

        // Note(qiduan): scaling is removed
        const float h_im = loc_h - 0.5;
        const float w_im = loc_w - 0.5;

        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w)
        {
          res += ms_deform_attn_im2col_bilinear(data_value_ptr, spatial_h, spatial_w, M, C, h_im, w_im, m, c) * weight;
        }

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
      }
      level_start_id += spatial_h * spatial_w;
      data_reference_ptr += Q << 1;
    }
    *data_col_ptr = res;
  }
}

void ms_deformable_im2col_gpu_kernel_wrapper(
  const float *data_value,
  const int64_t *data_spatial_shapes,
  const float *data_sampling_loc,
  const float *data_reference_points,
  const float *data_attn_weight,
  const int batch_size,
  const int spatial_size,
  const int num_heads,
  const int channels,
  const int num_levels,
  const int num_query,
  const int num_point,
  float *output,
  onnxruntime::Stream *device_stream)
{
  hipError_t err;
  int gpuCount = 0;
  if((err = hipGetDeviceCount(&gpuCount)) != hipSuccess){
    throw std::runtime_error(hipGetErrorString(err));
  }

  if(gpuCount <= 0){
    throw std::runtime_error("No CUDA compatible GPU available");
  }

  const int deviceID = 0;
  // Run on device 0
  if((err = hipSetDevice(deviceID)) != hipSuccess){
    throw std::runtime_error(hipGetErrorString(err));
  }

  hipDeviceProp_t property;
  if((err = hipGetDeviceProperties(&property, deviceID)) != hipSuccess){
    throw std::runtime_error(hipGetErrorString(err));
  }

  int num_output = batch_size * num_query * num_heads * channels;
  int num_threads = std::min(1024, property.maxThreadsPerBlock);
  int blocks = (num_output + num_threads - 1) / num_threads;

  hipStream_t stream = (device_stream == nullptr) ? static_cast<hipStream_t>(device_stream->GetHandle()) : nullptr;

  ms_deformable_im2col_gpu_kernel<<<blocks, num_threads, num_threads*3*sizeof(float), stream>>>(
    num_output,
    data_value,
    data_spatial_shapes,
    data_sampling_loc,
    data_reference_points,
    data_attn_weight,
    batch_size,
    spatial_size,
    num_heads,
    channels,
    num_levels,
    num_query,
    num_point,
    output
  );

  if((err = hipStreamSynchronize(stream)) != hipSuccess){
    throw std::runtime_error(hipGetErrorString(err));
  }
}
